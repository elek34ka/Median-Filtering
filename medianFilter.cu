#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "EasyBMP/EasyBMP.h"
#include "EasyBMP/EasyBMP_BMP.h"
#include <thrust/sort.h>

#include "MedianFilter.h"



__host__ void
medianFilterKernel(BMP &image, BMP &result_image, int width, int height) {

    int n = (offset+2)*(offset+2);
    float filter[n];
    RGBApixel colours[n];

    for (int i = offset; i < height - offset; i++) {
        for (int j = offset; j < width - offset; j++) {
            int count = 0;
            for (int x = -offset; x <= offset; x++)
                for (int y = -offset; y <= offset; y++) {
                    if (x + i < height && x + i >= 0 &&
                        y + j >= 0 && y + j < width) {

                        RGBApixel pixel = image.GetPixel(x + i, y + j);

                        float rgb_value =
                                (float) pixel.Red * 0.11f + (float) pixel.Green * 0.59f + (float) pixel.Blue * 0.3f;

                        filter[count] = rgb_value;
                        colours[count] = pixel;
                    }
                }

            thrust::sort_by_key(filter, filter + n, colours);

            result_image.SetPixel(i, j, colours[n/2]);

        }
    }
    result_image.SetBitDepth(32);
}
/*

__global__ void
medianFilterSharedKernel(unsigned char *inputImageKernel, unsigned char *outputImagekernel, int imageWidth,
                         int imageHeight) {
    //Set the row and col value for each thread.
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ unsigned char sharedmem[(TILE_SIZE + 2)][(TILE_SIZE + 2)];  //initialize shared memory
    //Take some values.
    bool is_x_left = (threadIdx.x == 0), is_x_right = (threadIdx.x == TILE_SIZE - 1);
    bool is_y_top = (threadIdx.y == 0), is_y_bottom = (threadIdx.y == TILE_SIZE - 1);

    //Initialize with zero
    if (is_x_left)
        sharedmem[threadIdx.x][threadIdx.y + 1] = 0;
    else if (is_x_right)
        sharedmem[threadIdx.x + 2][threadIdx.y + 1] = 0;
    if (is_y_top) {
        sharedmem[threadIdx.x + 1][threadIdx.y] = 0;
        if (is_x_left)
            sharedmem[threadIdx.x][threadIdx.y] = 0;
        else if (is_x_right)
            sharedmem[threadIdx.x + 2][threadIdx.y] = 0;
    } else if (is_y_bottom) {
        sharedmem[threadIdx.x + 1][threadIdx.y + 2] = 0;
        if (is_x_right)
            sharedmem[threadIdx.x + 2][threadIdx.y + 2] = 0;
        else if (is_x_left)
            sharedmem[threadIdx.x][threadIdx.y + 2] = 0;
    }

    //Setup pixel values
    sharedmem[threadIdx.x + 1][threadIdx.y + 1] = inputImageKernel[row * imageWidth + col];
    //Check for boundry conditions.
    if (is_x_left && (col > 0))
        sharedmem[threadIdx.x][threadIdx.y + 1] = inputImageKernel[row * imageWidth + (col - 1)];
    else if (is_x_right && (col < imageWidth - 1))
        sharedmem[threadIdx.x + 2][threadIdx.y + 1] = inputImageKernel[row * imageWidth + (col + 1)];
    if (is_y_top && (row > 0)) {
        sharedmem[threadIdx.x + 1][threadIdx.y] = inputImageKernel[(row - 1) * imageWidth + col];
        if (is_x_left)
            sharedmem[threadIdx.x][threadIdx.y] = inputImageKernel[(row - 1) * imageWidth + (col - 1)];
        else if (is_x_right)
            sharedmem[threadIdx.x + 2][threadIdx.y] = inputImageKernel[(row - 1) * imageWidth + (col + 1)];
    } else if (is_y_bottom && (row < imageHeight - 1)) {
        sharedmem[threadIdx.x + 1][threadIdx.y + 2] = inputImageKernel[(row + 1) * imageWidth + col];
        if (is_x_right)
            sharedmem[threadIdx.x + 2][threadIdx.y + 2] = inputImageKernel[(row + 1) * imageWidth + (col + 1)];
        else if (is_x_left)
            sharedmem[threadIdx.x][threadIdx.y + 2] = inputImageKernel[(row + 1) * imageWidth + (col - 1)];
    }

    __syncthreads();   //Wait for all threads to be done.

    //Setup the filter.
    unsigned char filterVector[9] = {sharedmem[threadIdx.x][threadIdx.y], sharedmem[threadIdx.x + 1][threadIdx.y],
                                     sharedmem[threadIdx.x + 2][threadIdx.y],
                                     sharedmem[threadIdx.x][threadIdx.y + 1],
                                     sharedmem[threadIdx.x + 1][threadIdx.y + 1],
                                     sharedmem[threadIdx.x + 2][threadIdx.y + 1],
                                     sharedmem[threadIdx.x][threadIdx.y + 2],
                                     sharedmem[threadIdx.x + 1][threadIdx.y + 2],
                                     sharedmem[threadIdx.x + 2][threadIdx.y + 2]};


    {
        for (int i = 0; i < 9; i++) {
            for (int j = i + 1; j < 9; j++) {
                if (filterVector[i] > filterVector[j]) {
                    //Swap Values.
                    char tmp = filterVector[i];
                    filterVector[i] = filterVector[j];
                    filterVector[j] = tmp;
                }
            }
        }
        outputImagekernel[row * imageWidth + col] = filterVector[4];   //Set the output image values.
    }
}
*/

bool MedianFilterGPU(BMP &image, BMP &result_image) {
    //Cuda error and image values.
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    int height = image.TellHeight();
    int width = image.TellWidth();

    medianFilterKernel(image, result_image, width, height);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float time = 0;
    hipEventElapsedTime(&time, start, stop);
    return true;
}
